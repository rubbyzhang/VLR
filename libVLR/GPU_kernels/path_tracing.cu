#include "hip/hip_runtime.h"
﻿#include "light_transport_common.cuh"

namespace VLR {
    // Common Closest Hit Program for All Primitive Types and Materials
    RT_PROGRAM void pathTracingIteration() {
        auto sbtr = optixu::getHitGroupSBTRecordData();
        const GeometryInstanceData &geomInst = plp.geomInstData[sbtr.geomInstData];

        Payload* payload;
        optixu::getPayloads<PayloadSignature>(&payload);

        KernelRNG &rng = payload->rng;
        WavelengthSamples &wls = payload->wls;

        SurfacePoint surfPt;
        float hypAreaPDF;
        calcSurfacePoint(geomInst, wls, &surfPt, &hypAreaPDF);

        const SurfaceMaterialDescriptor matDesc = plp.materialDescriptorBuffer[geomInst.materialIndex];
        BSDF bsdf(matDesc, surfPt, wls);
        EDF edf(matDesc, surfPt, wls);

        Vector3D dirOutLocal = surfPt.shadingFrame.toLocal(-asVector3D(optixGetWorldRayDirection()));

        // implicit light sampling
        SampledSpectrum spEmittance = edf.evaluateEmittance();
        if (spEmittance.hasNonZero()) {
            SampledSpectrum Le = spEmittance * edf.evaluate(EDFQuery(), dirOutLocal);

            float MISWeight = 1.0f;
            if (!payload->prevSampledType.isDelta() && payload->rayType != RayType::Primary) {
                float bsdfPDF = payload->prevDirPDF;
                float dist2 = surfPt.calcSquaredDistance(asPoint3D(optixGetWorldRayOrigin()));
                float lightPDF = geomInst.importance / getSumLightImportances() * hypAreaPDF * dist2 / std::fabs(dirOutLocal.z);
                MISWeight = (bsdfPDF * bsdfPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);
            }

            payload->contribution += payload->alpha * Le * MISWeight;
        }
        if (surfPt.atInfinity || payload->maxLengthTerminate)
            return;

        // Russian roulette
        float continueProb = std::fmin(payload->alpha.importance(wls.selectedLambdaIndex()) / payload->initImportance, 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb)
            return;
        payload->alpha /= continueProb;

        Normal3D geomNormalLocal = surfPt.shadingFrame.toLocal(surfPt.geometricNormal);
        BSDFQuery fsQuery(dirOutLocal, geomNormalLocal, DirectionType::All(), wls);

        // Next Event Estimation (explicit light sampling)
        if (bsdf.hasNonDelta()) {
            SurfaceLight light;
            float lightProb;
            float uPrim;
            selectSurfaceLight(rng.getFloat0cTo1o(), &light, &lightProb, &uPrim);

            SurfaceLightPosSample lpSample(uPrim, rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
            SurfaceLightPosQueryResult lpResult;
            light.sample(lpSample, &lpResult);

            const SurfaceMaterialDescriptor lightMatDesc = plp.materialDescriptorBuffer[lpResult.materialIndex];
            EDF ledf(lightMatDesc, lpResult.surfPt, wls);
            SampledSpectrum M = ledf.evaluateEmittance();

            Vector3D shadowRayDir;
            float squaredDistance;
            float fractionalVisibility;
            if (M.hasNonZero() && testVisibility(surfPt, lpResult.surfPt, wls, &shadowRayDir, &squaredDistance, &fractionalVisibility)) {
                Vector3D shadowRayDir_l = lpResult.surfPt.toLocal(-shadowRayDir);
                Vector3D shadowRayDir_sn = surfPt.toLocal(shadowRayDir);

                SampledSpectrum Le = M * ledf.evaluate(EDFQuery(), shadowRayDir_l);
                float lightPDF = lightProb * lpResult.areaPDF;

                SampledSpectrum fs = bsdf.evaluate(fsQuery, shadowRayDir_sn);
                float cosLight = lpResult.surfPt.calcCosTerm(-shadowRayDir);
                float bsdfPDF = bsdf.evaluatePDF(fsQuery, shadowRayDir_sn) * cosLight / squaredDistance;

                float MISWeight = 1.0f;
                if (!lpResult.posType.isDelta() && !std::isinf(lightPDF))
                    MISWeight = (lightPDF * lightPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);

                float G = fractionalVisibility * absDot(shadowRayDir_sn, geomNormalLocal) * cosLight / squaredDistance;
                float scalarCoeff = G * MISWeight / lightPDF; // 直接contributionの計算式に入れるとCUDAのバグなのかおかしな結果になる。
                payload->contribution += payload->alpha * Le * fs * scalarCoeff;
            }
        }

        BSDFSample sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        BSDFQueryResult fsResult;
        SampledSpectrum fs = bsdf.sample(fsQuery, sample, &fsResult);
        if (fs == SampledSpectrum::Zero() || fsResult.dirPDF == 0.0f)
            return;
        if (fsResult.sampledType.isDispersive() && !wls.singleIsSelected()) {
            fsResult.dirPDF /= SampledSpectrum::NumComponents();
            wls.setSingleIsSelected();
        }

        float cosFactor = dot(fsResult.dirLocal, geomNormalLocal);
        payload->alpha *= fs * (std::fabs(cosFactor) / fsResult.dirPDF);

        Vector3D dirIn = surfPt.fromLocal(fsResult.dirLocal);
        payload->origin = offsetRayOrigin(surfPt.position, cosFactor > 0.0f ? surfPt.geometricNormal : -surfPt.geometricNormal);
        payload->direction = dirIn;
        payload->prevDirPDF = fsResult.dirPDF;
        payload->prevSampledType = fsResult.sampledType;
        payload->terminate = false;
    }



    // JP: 本当は無限大の球のIntersection/Bounding Box Programを使用して環境光に関する処理もClosest Hit Programで統一的に行いたい。
    //     が、OptiXのBVHビルダーがLBVHベースなので無限大のAABBを生成するのは危険。
    //     仕方なくMiss Programで環境光を処理する。
    RT_PROGRAM void pathTracingMiss() {
        Payload* payload;
        optixu::getPayloads<PayloadSignature>(&payload);

        if (plp.envLightDescriptor.importance == 0)
            return;

        Vector3D direction = asVector3D(optixGetWorldRayDirection());
        float phi, theta;
        direction.toPolarYUp(&theta, &phi);

        float sinPhi, cosPhi;
        VLR::sincos(phi, &sinPhi, &cosPhi);
        Vector3D texCoord0Dir = normalize(Vector3D(-cosPhi, 0.0f, -sinPhi));
        ReferenceFrame shadingFrame;
        shadingFrame.x = texCoord0Dir;
        shadingFrame.z = -static_cast<Normal3D>(direction);
        shadingFrame.y = cross(shadingFrame.z, shadingFrame.x);

        SurfacePoint surfPt;
        surfPt.position = Point3D(direction.x, direction.y, direction.z);
        surfPt.shadingFrame = shadingFrame;
        surfPt.isPoint = false;
        surfPt.atInfinity = true;

        surfPt.geometricNormal = shadingFrame.z;
        surfPt.u = phi;
        surfPt.v = theta;
        phi += plp.envLightDescriptor.body.asInfSphere.rotationPhi;
        phi = phi - std::floor(phi / (2 * VLR_M_PI)) * 2 * VLR_M_PI;
        surfPt.texCoord = TexCoord2D(phi / (2 * VLR_M_PI), theta / VLR_M_PI);

        float hypAreaPDF = evaluateEnvironmentAreaPDF(phi, theta);

        const SurfaceMaterialDescriptor matDesc = plp.materialDescriptorBuffer[plp.envLightDescriptor.materialIndex];
        EDF edf(matDesc, surfPt, payload->wls);

        Vector3D dirOutLocal = surfPt.shadingFrame.toLocal(-direction);

        // implicit light sampling
        SampledSpectrum spEmittance = edf.evaluateEmittance();
        if (spEmittance.hasNonZero()) {
            SampledSpectrum Le = spEmittance * edf.evaluate(EDFQuery(), dirOutLocal);

            float MISWeight = 1.0f;
            if (!payload->prevSampledType.isDelta() && payload->rayType != RayType::Primary) {
                float bsdfPDF = payload->prevDirPDF;
                float dist2 = surfPt.calcSquaredDistance(asPoint3D(optixGetWorldRayOrigin()));
                float lightPDF = plp.envLightDescriptor.importance / getSumLightImportances() * hypAreaPDF * dist2 / std::fabs(dirOutLocal.z);
                MISWeight = (bsdfPDF * bsdfPDF) / (lightPDF * lightPDF + bsdfPDF * bsdfPDF);
            }

            payload->contribution += payload->alpha * Le * MISWeight;
        }
    }



    // Common Ray Generation Program for All Camera Types
    RT_PROGRAM void pathTracing() {
        uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

        KernelRNG rng = plp.rngBuffer[launchIndex];

        float2 p = make_float2(launchIndex.x + rng.getFloat0cTo1o(),
                               launchIndex.y + rng.getFloat0cTo1o());

        float selectWLPDF;
        WavelengthSamples wls = WavelengthSamples::createWithEqualOffsets(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), &selectWLPDF);

        LensPosSample We0Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        LensPosQueryResult We0Result;
        SampledSpectrum We0 = plp.progSampleLensPosition(wls, We0Sample, &We0Result);

        IDFSample We1Sample(p.x / plp.imageSize.x, p.y / plp.imageSize.y);
        IDFQueryResult We1Result;
        SampledSpectrum We1 = plp.progSampleIDF(We0Result.surfPt, wls, We1Sample, &We1Result);

        Point3D rayOrg = We0Result.surfPt.position;
        Vector3D rayDir = We0Result.surfPt.fromLocal(We1Result.dirLocal);
        RayType rayType = RayType::Primary;
        SampledSpectrum alpha = (We0 * We1) * (We0Result.surfPt.calcCosTerm(rayDir) / (We0Result.areaPDF * We1Result.dirPDF * selectWLPDF));

        Payload payload;
        payload.maxLengthTerminate = false;
        payload.rng = rng;
        payload.initImportance = alpha.importance(wls.selectedLambdaIndex());
        payload.wls = wls;
        payload.alpha = alpha;
        payload.contribution = SampledSpectrum::Zero();
        Payload* payloadPtr = &payload;

        const uint32_t MaxPathLength = 25;
        uint32_t pathLength = 0;
        while (true) {
            payload.rayType = rayType;
            payload.terminate = true;
            ++pathLength;
            if (pathLength >= MaxPathLength)
                payload.maxLengthTerminate = true;
            optixu::trace<PayloadSignature>(
                plp.topGroup,
                asOptiXType(rayOrg), asOptiXType(rayDir),
                0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
                rayType, RayType::NumTypes, rayType,
                payloadPtr);

            if (payload.terminate)
                break;
            VLRAssert(pathLength < MaxPathLength, "Path should be terminated... Something went wrong...");

            rayOrg = payload.origin;
            rayDir = payload.direction;
            rayType = RayType::Scattered;
        }
        plp.rngBuffer[launchIndex] = payload.rng;
        if (!payload.contribution.allFinite()) {
            vlrprintf("Pass %u, (%u, %u): Not a finite value.\n", plp.numAccumFrames, launchIndex.x, launchIndex.y);
            return;
        }

        if (plp.numAccumFrames == 1)
            plp.outputBuffer[launchIndex].reset();
        plp.outputBuffer[launchIndex].add(wls, payload.contribution);
    }



    //// Exception Program
    //RT_PROGRAM void exception() {
    //    //uint32_t code = rtGetExceptionCode();
    //    rtPrintExceptionDetails();
    //}
}
