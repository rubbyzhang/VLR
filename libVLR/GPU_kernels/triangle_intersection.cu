#include "hip/hip_runtime.h"
﻿#include "kernel_common.cuh"

namespace VLR {
    RT_FUNCTION bool intersectTriangle(const Point3D &rayOrg, const Vector3D &rayDir, float tMin, float tMax,
                                       const Point3D &p0, const Point3D &p1, const Point3D &p2,
                                       Normal3D* n, float* t, float* b1, float* b2) {
        Vector3D e0 = p1 - p0;
        Vector3D e1 = p0 - p2;
        *n = static_cast<Normal3D>(cross(e1, e0));

        const Vector3D e2 = (1.0f / dot(*n, rayDir)) * (p0 - rayOrg);
        const Vector3D i = cross(rayDir, e2);

        *b1 = dot(i, e1);
        *b2 = dot(i, e0);
        *t = dot(*n, e2);

        return ((*t < tMax) & (*t > tMin) & (*b1 >= 0.0f) & (*b2 >= 0.0f) & (*b1 + *b2 <= 1));
    }

    // Intersection Program
    RT_PROGRAM void RT_IS_NAME(intersectTriangle)() {
        auto sbtr = optixu::getHitGroupSBTRecordData();
        const GeometryInstanceData &geomInstData = plp.geomInstData[sbtr.geomInstData];
        uint32_t primIndex = optixGetPrimitiveIndex();

        const Triangle &triangle = geomInstData.triangleBuffer[primIndex];
        const Vertex &v0 = geomInstData.vertexBuffer[triangle.index0];
        const Vertex &v1 = geomInstData.vertexBuffer[triangle.index1];
        const Vertex &v2 = geomInstData.vertexBuffer[triangle.index2];

        Point3D rayOrg = asPoint3D(optixGetObjectRayOrigin());
        Vector3D rayDir = asVector3D(optixGetObjectRayDirection());
        float tMin = optixGetRayTmin();
        float tMax = optixGetRayTmax();

        // use a triangle intersection function defined in optix_math_namespace.h
        Normal3D gn;
        float t;
        float b0, b1, b2;
        if (!intersectTriangle(rayOrg, rayDir, tMin, tMax,
                               v0.position, v1.position, v2.position,
                               &gn, &t, &b1, &b2))
            return;

        b0 = 1.0f - b1 - b2;

        optixu::reportIntersection(t, 0, b0, b1);
    }

    //// Bounding Box Program
    //RT_PROGRAM void calcBBoxForTriangle(int32_t primIdx, float result[6]) {
    //    const Triangle &triangle = pv_triangleBuffer[primIdx];
    //    const Point3D &p0 = pv_vertexBuffer[triangle.index0].position;
    //    const Point3D &p1 = pv_vertexBuffer[triangle.index1].position;
    //    const Point3D &p2 = pv_vertexBuffer[triangle.index2].position;

    //    //optix::Aabb* bbox = (optix::Aabb*)result;
    //    //*bbox = optix::Aabb(asOptiXType(p0), asOptiXType(p1), asOptiXType(p2));

    //    BoundingBox3D* bbox = (BoundingBox3D*)result;
    //    *bbox = BoundingBox3D(Point3D(INFINITY), Point3D(-INFINITY));
    //    bbox->unify(p0);
    //    bbox->unify(p1);
    //    bbox->unify(p2);
    //}

    //// Attribute Program (for GeometryTriangles)
    //RT_PROGRAM void calcAttributeForTriangle() {
    //    optix::float2 bc = rtGetTriangleBarycentrics();
    //    a_hitPointParam.b0 = 1 - bc.x - bc.y;
    //    a_hitPointParam.b1 = bc.x;
    //    a_hitPointParam.primIndex = rtGetPrimitiveIndex();
    //}



    RT_CALLABLE_PROGRAM void RT_DC_NAME(decodeHitPointForTriangle)(const HitPointParameter &param, SurfacePoint* surfPt, float* hypAreaPDF) {
        auto sbtr = optixu::getHitGroupSBTRecordData();
        const GeometryInstanceData &geomInstData = plp.geomInstData[sbtr.geomInstData];

        const Triangle &triangle = geomInstData.triangleBuffer[param.primIndex];
        const Vertex &v0 = geomInstData.vertexBuffer[triangle.index0];
        const Vertex &v1 = geomInstData.vertexBuffer[triangle.index1];
        const Vertex &v2 = geomInstData.vertexBuffer[triangle.index2];

        Vector3D e1 = transform<TransformKind::ObjectToWorld>(v1.position - v0.position);
        Vector3D e2 = transform<TransformKind::ObjectToWorld>(v2.position - v0.position);
        Normal3D geometricNormal = static_cast<Normal3D>(cross(e1, e2));
        float area = geometricNormal.length() / 2; // TODO: スケーリングの考慮。
        geometricNormal /= 2 * area;

        // JP: プログラムがこの点を光源としてサンプルする場合の面積に関する(仮想的な)PDFを求める。
        // EN: calculate a hypothetical area PDF value in the case where the program sample this point as light.
        float probLightPrim = area / geomInstData.sumImportances;
        *hypAreaPDF = probLightPrim / area;

        float b0 = param.b0, b1 = param.b1, b2 = 1.0f - param.b0 - param.b1;
        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = transform<TransformKind::ObjectToWorld>(position);
        shadingNormal = normalize(transform<TransformKind::ObjectToWorld>(shadingNormal));
        tc0Direction = transform<TransformKind::ObjectToWorld>(tc0Direction);

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * static_cast<Vector3D>(shadingNormal));

        surfPt->position = position;
        surfPt->shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt->isPoint = false;
        surfPt->atInfinity = false;
        surfPt->geometricNormal = geometricNormal;
        surfPt->u = b0;
        surfPt->v = b1;
        surfPt->texCoord = texCoord;
    }



    RT_CALLABLE_PROGRAM void RT_DC_NAME(sampleTriangleMesh)(const GeometryInstanceDescriptor::Body &desc, const SurfaceLightPosSample &sample, SurfaceLightPosQueryResult* result) {
        float primProb;
        uint32_t primIdx = desc.asTriMesh.primDistribution.sample(sample.uElem, &primProb);

        const Triangle &triangle = desc.asTriMesh.triangleBuffer[primIdx];
        const Vertex &v0 = desc.asTriMesh.vertexBuffer[triangle.index0];
        const Vertex &v1 = desc.asTriMesh.vertexBuffer[triangle.index1];
        const Vertex &v2 = desc.asTriMesh.vertexBuffer[triangle.index2];

        StaticTransform transform = desc.asTriMesh.transform;

        Vector3D e1 = transform * (v1.position - v0.position);
        Vector3D e2 = transform * (v2.position - v0.position);
        Normal3D geometricNormal = static_cast<Normal3D>(cross(e1, e2));
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        result->areaPDF = primProb / area;
        result->posType = DirectionType::Emission() | DirectionType::LowFreq();

        float b0, b1, b2;
        uniformSampleTriangle(sample.uPos[0], sample.uPos[1], &b0, &b1);
        b2 = 1.0f - b0 - b1;

        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = transform * position;
        shadingNormal = normalize(transform * shadingNormal);
        tc0Direction = transform * tc0Direction;

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * static_cast<Vector3D>(shadingNormal));

        SurfacePoint &surfPt = result->surfPt;

        surfPt.position = position;
        surfPt.shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt.isPoint = false;
        surfPt.atInfinity = false;
        surfPt.geometricNormal = geometricNormal;
        surfPt.u = b0;
        surfPt.v = b1;
        surfPt.texCoord = texCoord;
    }
}
